
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

/* Bounds of the Mandelbrot set */
#define X_MIN -1.78
#define X_MAX 0.78
#define Y_MIN -0.961
#define Y_MAX 0.961

typedef struct {

  int nb_rows, nb_columns; /* Dimensions */
  char * pixels; /* Linearized matrix of pixels */

} Image;

static void error_options () {

  fprintf (stderr, "Use : ./mandel [options]\n\n");
  fprintf (stderr, "Options \t Meaning \t\t Default val.\n\n");
  fprintf (stderr, "-n \t\t Nb iter. \t\t 100\n");
  fprintf (stderr, "-b \t\t Bounds \t\t -1.78 0.78 -0.961 0.961\n");
  fprintf (stderr, "-d \t\t Dimensions \t\t 1024 768\n");
  fprintf (stderr, "-f \t\t File \t\t /tmp/mandel.ppm\n");
  exit (1);
}

static void analyzis (int argc, char * * argv, int * nb_iter, double * x_min, double * x_max, double * y_min, double * y_max, int * width, int * height, char * * path) {

  const char * opt = "b:d:n:f:" ;
  int c ;

  /* Default values */
  * nb_iter = 500;
  * x_min = X_MIN;
  * x_max = X_MAX;
  * y_min = Y_MIN;
  * y_max = Y_MAX;
  * width = 1024;
  * height = 768;
  * path = "tmp/mandel.ppm";

  /* Analysis of arguments */
  while ((c = getopt (argc, argv, opt)) != EOF) {
    
    switch (c) {
      
    case 'b':
      sscanf (optarg, "%lf", x_min);
      sscanf (argv [optind ++], "%lf", x_max);
      sscanf (argv [optind ++], "%lf", y_min);
      sscanf (argv [optind ++], "%lf", y_max);
      break ;
    case 'd': /* width */
      sscanf (optarg, "%d", width);
      sscanf (argv [optind ++], "%d", height);
      break;
    case 'n': /* Number of iterations */
      * nb_iter = atoi (optarg);
      break;
    case 'f': /* Output file */
      * path = optarg;
      break;
    default :
      error_options ();
    };
  }  
}

static void initialization (Image * im, int nb_columns, int nb_rows) {
  im -> nb_rows = nb_rows;
  im -> nb_columns = nb_columns;
  im -> pixels = (char *) malloc (sizeof (char) * nb_rows * nb_columns); /* Space memory allocation */
} 

static void save (const Image * im, const char * path) {
  /* Image saving using the ASCII format'.PPM' */
  unsigned i;
  FILE * f = fopen (path, "w");  
  fprintf (f, "P6\n%d %d\n255\n", im -> nb_columns, im -> nb_rows); 
  for (i = 0; i < im -> nb_columns * im -> nb_rows; i ++) {
    char c = im -> pixels [i];
    fprintf (f, "%c%c%c", c, c, c); /* Monochrome weight */
  }
  fclose (f);
}

__global__ void kercud(double dx, double dy, char * pixels, int nb_iter, double x_min, double y_max, int num_col){
	   int index_of_X = blockIdx.x * blockDim.x + threadIdx.x;
	   int index_of_Y = blockIdx.y * blockDim.y + threadIdx.y;

	   double a = x_min + index_of_Y *dx, b = y_max - index_of_X * dy, x = 0, y = 0;
	   int i = 0;
	   while (i < nb_iter){
	   	 double tmp = x;
		 x = x * x - y * y + a;
		 y = 2 * tmp * y + b;
		 if (x * x + y * y > 4){
		    break;
		 }
		 else {
		      i ++;
		 }
	   }
	   pixels [index_of_X * num_col + index_of_Y]= (double) i / nb_iter * 255; //formula instead of pos
}

static void compute (Image * im, int nb_iter, double x_min, double x_max, double y_min, double y_max) {
    
  double dx = (x_max - x_min) / im -> nb_columns, dy = (y_max - y_min) / im -> nb_rows; /* Discretization */
  int row_num = im -> nb_rows, num_col = im -> nb_columns;

  dim3 size_of_block(16,16,1);
  dim3 no_of_thrds_in_block(row_num/16, num_col/16, 1);

  char * cuda_pixel;
  hipMalloc(&cuda_pixel, sizeof(char) * row_num * num_col);
  hipMemcpy(cuda_pixel, im -> pixels, sizeof(char) * row_num * num_col, hipMemcpyHostToDevice);

  kercud <<< no_of_thrds_in_block, size_of_block >>> (dx, dy, cuda_pixel, nb_iter, x_min, y_max, num_col);
  hipDeviceSynchronize();

  hipMemcpy(im -> pixels, cuda_pixel, sizeof(char) * row_num * num_col, hipMemcpyDeviceToHost);
  hipFree(cuda_pixel);
}

int main (int argc, char * * argv) {
  
  int nb_iter, width, height; /* Degree of precision, dimensions of the image */  
  double x_min, x_max, y_min, y_max; /* Bounds of representation */
  char * path; /* File destination */
  Image im;
  analyzis(argc, argv, & nb_iter, & x_min, & x_max, & y_min, & y_max, & width, & height, & path);
  initialization (& im, width, height);
  compute (& im, nb_iter, x_min, x_max, y_min, y_max);
  save (& im, path);

  return 0 ;
}
